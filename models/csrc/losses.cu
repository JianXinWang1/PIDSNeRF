#include "hip/hip_runtime.h"
#include "utils.h"
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

float pi = 3.1415926;

// for details of the formulae, please see https://arxiv.org/pdf/2206.05085.pdf

template <typename scalar_t>
__global__ void prefix_sums_kernel(
    const scalar_t* __restrict__ ws,
    const scalar_t* __restrict__ wts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    scalar_t* __restrict__ ws_inclusive_scan,
    scalar_t* __restrict__ ws_exclusive_scan,
    scalar_t* __restrict__ wts_inclusive_scan,
    scalar_t* __restrict__ wts_exclusive_scan
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // compute prefix sum of ws and ws*ts
    // [a0, a1, a2, a3, ...] -> [a0, a0+a1, a0+a1+a2, a0+a1+a2+a3, ...]
    thrust::inclusive_scan(thrust::device,
                           ws+start_idx,
                           ws+start_idx+N_samples,
                           ws_inclusive_scan+start_idx);
    thrust::inclusive_scan(thrust::device,
                           wts+start_idx,
                           wts+start_idx+N_samples,
                           wts_inclusive_scan+start_idx);
    // [a0, a1, a2, a3, ...] -> [0, a0, a0+a1, a0+a1+a2, ...]
    thrust::exclusive_scan(thrust::device,
                           ws+start_idx,
                           ws+start_idx+N_samples,
                           ws_exclusive_scan+start_idx);
    thrust::exclusive_scan(thrust::device,
                           wts+start_idx,
                           wts+start_idx+N_samples,
                           wts_exclusive_scan+start_idx);
}


template <typename scalar_t>
__global__ void distortion_loss_fw_kernel(
    const scalar_t* __restrict__ _loss,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> loss
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    loss[ray_idx] = thrust::reduce(thrust::device, 
                                   _loss+start_idx,
                                   _loss+start_idx+N_samples,
                                   (scalar_t)0);
}


std::vector<torch::Tensor> distortion_loss_fw_cu(
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a
){
    const int N_rays = rays_a.size(0), N = ws.size(0);

    auto wts = ws * ts;

    auto ws_inclusive_scan = torch::zeros({N}, ws.options());
    auto ws_exclusive_scan = torch::zeros({N}, ws.options());
    auto wts_inclusive_scan = torch::zeros({N}, ws.options());
    auto wts_exclusive_scan = torch::zeros({N}, ws.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_fw_cu_prefix_sums", 
    ([&] {
        prefix_sums_kernel<scalar_t><<<blocks, threads>>>(
            ws.data_ptr<scalar_t>(),
            wts.data_ptr<scalar_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            ws_inclusive_scan.data_ptr<scalar_t>(),
            ws_exclusive_scan.data_ptr<scalar_t>(),
            wts_inclusive_scan.data_ptr<scalar_t>(),
            wts_exclusive_scan.data_ptr<scalar_t>()
        );
    }));

    auto _loss = 2*(wts_inclusive_scan*ws_exclusive_scan-
                    ws_inclusive_scan*wts_exclusive_scan) + 1.0f/3*ws*ws*deltas;

    auto loss = torch::zeros({N_rays}, ws.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_fw_cu", 
    ([&] {
        distortion_loss_fw_kernel<scalar_t><<<blocks, threads>>>(
            _loss.data_ptr<scalar_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            loss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {loss, ws_inclusive_scan, wts_inclusive_scan};
}


template <typename scalar_t>
__global__ void distortion_loss_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dloss,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws_inclusive_scan,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> wts_inclusive_scan,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dws,
    const torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> rgb_len
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];
    const int end_idx = start_idx+N_samples-1;

    const scalar_t ws_sum = ws_inclusive_scan[end_idx];
    const scalar_t wts_sum = wts_inclusive_scan[end_idx];
    // fill in dL_dws from start_idx to end_idx

    if(ray_idx < rgb_len[0])
    {
        for (int s=start_idx; s<=end_idx; s++){
        dL_dws[s] = dL_dloss[ray_idx] * 2 * (
            (s==start_idx?
                (scalar_t)0:
                (ts[s]*ws_inclusive_scan[s-1]-wts_inclusive_scan[s-1])
            ) + 
            (wts_sum-wts_inclusive_scan[s]-ts[s]*(ws_sum-ws_inclusive_scan[s]))
        );
        dL_dws[s] += dL_dloss[ray_idx] * (scalar_t)2/3*ws[s]*deltas[s];
        }
    }
}


torch::Tensor distortion_loss_bw_cu(
    const torch::Tensor dL_dloss,
    const torch::Tensor ws_inclusive_scan,
    const torch::Tensor wts_inclusive_scan,
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const torch::Tensor rgb_len
){
    const int N_rays = rays_a.size(0), N = ws.size(0);

    auto dL_dws = torch::zeros({N}, dL_dloss.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_bw_cu", 
    ([&] {
        distortion_loss_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dloss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws_inclusive_scan.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            wts_inclusive_scan.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            dL_dws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rgb_len.packed_accessor<int64_t, 1, torch::RestrictPtrTraits>()
        );
    }));

    return dL_dws;
}






template <typename scalar_t>
__global__ void depth_loss_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> detals,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> target,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> loss
    
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];
    if(ray_idx>=1024){
        int depth_idx = ray_idx - 1024;
        float loss_sum = 0;

        int samples = 0;
        float mean_value = target[depth_idx];
        float judge = 0;

        float sigma = 0.001;
        
        while (samples < N_samples) {
            const int s = start_idx + samples;
            float t = ts[s];
            float dt = detals[s];

            float A = 1/(sigma * sqrt(2*M_PI));


            // DS KL
            float B1 = (-1.0/2) * (((t-mean_value)/sigma) * ((t-mean_value)/sigma));
            float C1 = exp(B1);
 
            

            judge += ws[s] * t;

            loss_sum += -log(ws[s]+0.0001)*A*C1 * dt;

            
            samples++;    
        }
        if(judge>mean_value-3*sigma && judge<mean_value+3*sigma){
            loss[ray_idx]=0;
        }
        else{
            loss[ray_idx] = loss_sum;
        }
            
    }
}


torch::Tensor depth_loss_fw_cu(
    const torch::Tensor ws,
    const torch::Tensor ts,
    const torch::Tensor detals,
    const torch::Tensor depth,
    const torch::Tensor target,
    const torch::Tensor rays_a
    
){
    const int N_rays = rays_a.size(0);
    auto loss = torch::zeros({N_rays}, depth.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;


    AT_DISPATCH_FLOATING_TYPES_AND_HALF(depth.type(), "depth_loss_fw_cu", 
    ([&] {
        depth_loss_fw_kernel<scalar_t><<<blocks, threads>>>(
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            detals.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            target.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            loss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return loss;
}









template <typename scalar_t>
__global__ void depth_loss_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dloss,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> target,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> detals,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dws
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];
    const int end_idx = start_idx+N_samples;

    // fill in dL_dws from start_idx to end_idx

    if (ray_idx>=1024){
        for (int s=start_idx; s<end_idx; s++){
            float tem1 = (ts[s]-target[ray_idx-1024]) * (ts[s]-target[ray_idx-1024]);
            dL_dws[s] = -dL_dloss[ray_idx]* (1/(ws[s]+0.0001))*exp(-tem1/(2*0.001)) *detals[s];
        }
    }
    //  for (int s=start_idx; s<end_idx; s++){
    //     dL_dws[s] = dL_dloss[ray_idx]* 2*(depth[ray_idx] - target[ray_idx]) *ts[s];
    // }
    
}




torch::Tensor depth_loss_bw_cu(
    const torch::Tensor dL_dloss,
    const torch::Tensor ts,
    const torch::Tensor depth,
    const torch::Tensor target,
    const torch::Tensor rays_a,
    const torch::Tensor ws,
    const torch::Tensor detals
){
    const int N_rays = rays_a.size(0), N = ts.size(0);

    auto dL_dws = torch::zeros({N}, dL_dloss.options());


    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ts.type(), "depth_loss_bw_cu", 
    ([&] {
        depth_loss_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dloss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            target.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            detals.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_dws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return dL_dws;
}




